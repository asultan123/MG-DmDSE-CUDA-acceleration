#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <sys/time.h>
#include "cudaDmy.cuh"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>
#include <fstream>
#include <map>
#include <set> 
#include <string>
#include <vector>
#include <iterator>
#include <algorithm>
#include <bits/stdc++.h> 
#include <assert.h>
#include <limits>
#include <string>
#include <sstream>
#include <chrono> 

// #define DEBUG

using std::ifstream;
using std::cout;
using std::endl;
using std::map;
using std::set;
using std::pair;
using std::make_pair;
using std::string;
using std::vector;
using std::advance;
using std::sort;
using std::accumulate;
using std::max_element;
using namespace std::chrono; 

unsigned int fileInfoArray[1047][3] = {{0, 0, 0},{0, 1, 0},{0, 2, 0},{0, 3, 0},{0, 4, 0},{0, 5, 0},{0, 6, 0},{0, 7, 0},{0, 8, 0},{0, 9, 0},{0, 10, 0},{0, 11, 0},{0, 12, 0},{0, 13, 0},{0, 14, 0},{0, 15, 0},{0, 16, 0},{0, 17, 0},{0, 18, 0},{0, 19, 0},{0, 20, 0},{0, 21, 0},{0, 22, 0},{0, 23, 0},{0, 24, 0},{0, 25, 0},{0, 26, 0},{0, 27, 0},{0, 28, 0},{0, 29, 0},{0, 30, 0},{0, 31, 0},{0, 32, 0},{0, 33, 0},{0, 34, 0},{0, 35, 0},{0, 36, 0},{0, 37, 0},{0, 38, 0},{0, 39, 0},{1, 0, 0},{1, 0, 1},{1, 1, 0},{1, 2, 0},{1, 2, 1},{1, 3, 0},{1, 4, 0},{1, 4, 1},{1, 5, 0},{1, 5, 1},{1, 5, 2},{1, 5, 3},{1, 6, 0},{1, 6, 1},{1, 7, 0},{1, 8, 0},{1, 9, 0},{1, 9, 1},{1, 10, 0},{1, 10, 1},{1, 11, 0},{1, 12, 0},{1, 13, 0},{1, 13, 1},{1, 14, 0},{1, 14, 1},{1, 14, 2},{1, 14, 3},{1, 15, 0},{1, 16, 0},{1, 17, 0},{1, 18, 0},{1, 19, 0},{1, 19, 1},{1, 20, 0},{1, 21, 0},{1, 22, 0},{1, 23, 0},{1, 24, 0},{1, 25, 0},{1, 25, 1},{1, 26, 0},{1, 27, 0},{1, 27, 1},{1, 28, 0},{1, 28, 1},{1, 29, 0},{1, 29, 1},{1, 29, 2},{1, 29, 3},{1, 30, 0},{1, 31, 0},{1, 32, 0},{1, 32, 1},{1, 32, 2},{1, 32, 3},{1, 32, 4},{1, 32, 5},{1, 32, 6},{1, 32, 7},{1, 33, 0},{1, 34, 0},{1, 35, 0},{1, 35, 1},{1, 35, 2},{1, 35, 3},{1, 36, 0},{1, 37, 0},{1, 37, 1},{1, 38, 0},{1, 38, 1},{1, 39, 0},{1, 39, 1},{2, 0, 0},{2, 1, 0},{2, 1, 1},{2, 1, 2},{2, 1, 3},{2, 1, 4},{2, 1, 5},{2, 1, 6},{2, 1, 7},{2, 1, 8},{2, 1, 9},{2, 1, 10},{2, 1, 11},{2, 1, 12},{2, 1, 13},{2, 1, 14},{2, 1, 15},{2, 2, 0},{2, 3, 0},{2, 3, 1},{2, 3, 2},{2, 3, 3},{2, 4, 0},{2, 4, 1},{2, 4, 2},{2, 4, 3},{2, 4, 4},{2, 4, 5},{2, 4, 6},{2, 4, 7},{2, 5, 0},{2, 5, 1},{2, 6, 0},{2, 6, 1},{2, 6, 2},{2, 6, 3},{2, 6, 4},{2, 6, 5},{2, 6, 6},{2, 6, 7},{2, 7, 0},{2, 7, 1},{2, 7, 2},{2, 7, 3},{2, 7, 4},{2, 7, 5},{2, 7, 6},{2, 7, 7},{2, 7, 8},{2, 7, 9},{2, 7, 10},{2, 7, 11},{2, 7, 12},{2, 7, 13},{2, 7, 14},{2, 7, 15},{2, 8, 0},{2, 9, 0},{2, 9, 1},{2, 10, 0},{2, 10, 1},{2, 10, 2},{2, 10, 3},{2, 10, 4},{2, 10, 5},{2, 10, 6},{2, 10, 7},{2, 11, 0},{2, 11, 1},{2, 11, 2},{2, 11, 3},{2, 11, 4},{2, 11, 5},{2, 11, 6},{2, 11, 7},{2, 12, 0},{2, 12, 1},{2, 13, 0},{2, 13, 1},{2, 13, 2},{2, 13, 3},{2, 13, 4},{2, 13, 5},{2, 13, 6},{2, 13, 7},{2, 14, 0},{2, 14, 1},{2, 15, 0},{2, 16, 0},{2, 16, 1},{2, 16, 2},{2, 16, 3},{2, 16, 4},{2, 16, 5},{2, 16, 6},{2, 16, 7},{2, 16, 8},{2, 16, 9},{2, 16, 10},{2, 16, 11},{2, 16, 12},{2, 16, 13},{2, 16, 14},{2, 16, 15},{2, 16, 16},{2, 16, 17},{2, 16, 18},{2, 16, 19},{2, 16, 20},{2, 16, 21},{2, 16, 22},{2, 16, 23},{2, 16, 24},{2, 16, 25},{2, 16, 26},{2, 16, 27},{2, 16, 28},{2, 16, 29},{2, 16, 30},{2, 16, 31},{2, 16, 32},{2, 16, 33},{2, 16, 34},{2, 16, 35},{2, 16, 36},{2, 16, 37},{2, 16, 38},{2, 16, 39},{2, 16, 40},{2, 16, 41},{2, 16, 42},{2, 16, 43},{2, 16, 44},{2, 16, 45},{2, 16, 46},{2, 16, 47},{2, 16, 48},{2, 16, 49},{2, 16, 50},{2, 16, 51},{2, 16, 52},{2, 16, 53},{2, 16, 54},{2, 16, 55},{2, 16, 56},{2, 16, 57},{2, 16, 58},{2, 16, 59},{2, 16, 60},{2, 16, 61},{2, 16, 62},{2, 16, 63},{2, 17, 0},{2, 17, 1},{2, 18, 0},{2, 18, 1},{2, 19, 0},{2, 20, 0},{2, 20, 1},{2, 20, 2},{2, 20, 3},{2, 21, 0},{2, 21, 1},{2, 21, 2},{2, 21, 3},{2, 22, 0},{2, 22, 1},{2, 22, 2},{2, 22, 3},{2, 22, 4},{2, 22, 5},{2, 22, 6},{2, 22, 7},{2, 23, 0},{2, 23, 1},{2, 24, 0},{2, 24, 1},{2, 25, 0},{2, 25, 1},{2, 25, 2},{2, 25, 3},{2, 25, 4},{2, 25, 5},{2, 25, 6},{2, 25, 7},{2, 25, 8},{2, 25, 9},{2, 25, 10},{2, 25, 11},{2, 25, 12},{2, 25, 13},{2, 25, 14},{2, 25, 15},{2, 26, 0},{2, 27, 0},{2, 27, 1},{2, 28, 0},{2, 28, 1},{2, 28, 2},{2, 28, 3},{2, 28, 4},{2, 28, 5},{2, 28, 6},{2, 28, 7},{2, 29, 0},{2, 29, 1},{2, 29, 2},{2, 29, 3},{2, 30, 0},{2, 31, 0},{2, 32, 0},{2, 33, 0},{2, 33, 1},{2, 34, 0},{2, 35, 0},{2, 35, 1},{2, 35, 2},{2, 35, 3},{2, 36, 0},{2, 37, 0},{2, 38, 0},{2, 38, 1},{2, 38, 2},{2, 38, 3},{2, 38, 4},{2, 38, 5},{2, 38, 6},{2, 38, 7},{2, 39, 0},{2, 39, 1},{2, 39, 2},{2, 39, 3},{2, 39, 4},{2, 39, 5},{2, 39, 6},{2, 39, 7},{3, 0, 0},{3, 0, 1},{3, 1, 0},{3, 1, 1},{3, 1, 2},{3, 1, 3},{3, 1, 4},{3, 1, 5},{3, 1, 6},{3, 1, 7},{3, 1, 8},{3, 1, 9},{3, 1, 10},{3, 1, 11},{3, 1, 12},{3, 1, 13},{3, 1, 14},{3, 1, 15},{3, 2, 0},{3, 2, 1},{3, 3, 0},{3, 3, 1},{3, 3, 2},{3, 3, 3},{3, 4, 0},{3, 4, 1},{3, 4, 2},{3, 4, 3},{3, 4, 4},{3, 4, 5},{3, 4, 6},{3, 4, 7},{3, 4, 8},{3, 4, 9},{3, 4, 10},{3, 4, 11},{3, 4, 12},{3, 4, 13},{3, 4, 14},{3, 4, 15},{3, 5, 0},{3, 5, 1},{3, 5, 2},{3, 5, 3},{3, 6, 0},{3, 6, 1},{3, 6, 2},{3, 6, 3},{3, 6, 4},{3, 6, 5},{3, 6, 6},{3, 6, 7},{3, 6, 8},{3, 6, 9},{3, 6, 10},{3, 6, 11},{3, 6, 12},{3, 6, 13},{3, 6, 14},{3, 6, 15},{3, 7, 0},{3, 7, 1},{3, 7, 1},{3, 7, 2},{3, 7, 3},{3, 7, 4},{3, 7, 5},{3, 7, 6},{3, 7, 7},{3, 7, 8},{3, 7, 9},{3, 7, 10},{3, 7, 11},{3, 7, 13},{3, 7, 14},{3, 7, 15},{3, 7, 16},{3, 7, 17},{3, 7, 18},{3, 7, 19},{3, 7, 20},{3, 7, 21},{3, 7, 22},{3, 7, 23},{3, 7, 24},{3, 7, 25},{3, 7, 26},{3, 7, 27},{3, 7, 28},{3, 7, 29},{3, 7, 30},{3, 7, 31},{3, 8, 0},{3, 9, 0},{3, 9, 1},{3, 9, 2},{3, 9, 3},{3, 10, 0},{3, 10, 1},{3, 10, 2},{3, 10, 3},{3, 10, 4},{3, 10, 5},{3, 10, 6},{3, 10, 7},{3, 10, 8},{3, 10, 9},{3, 10, 10},{3, 10, 11},{3, 10, 12},{3, 10, 13},{3, 10, 14},{3, 10, 15},{3, 11, 0},{3, 11, 1},{3, 11, 2},{3, 11, 3},{3, 11, 4},{3, 11, 5},{3, 11, 6},{3, 11, 7},{3, 11, 8},{3, 11, 9},{3, 11, 10},{3, 11, 11},{3, 11, 12},{3, 11, 13},{3, 11, 14},{3, 11, 15},{3, 12, 0},{3, 12, 1},{3, 12, 2},{3, 12, 3},{3, 13, 0},{3, 13, 1},{3, 13, 2},{3, 13, 3},{3, 13, 4},{3, 13, 5},{3, 13, 6},{3, 13, 7},{3, 13, 8},{3, 13, 9},{3, 13, 10},{3, 13, 11},{3, 13, 12},{3, 13, 13},{3, 13, 14},{3, 13, 15},{3, 14, 0},{3, 14, 1},{3, 14, 2},{3, 14, 3},{3, 15, 0},{3, 16, 0},{3, 16, 1},{3, 16, 2},{3, 16, 3},{3, 16, 4},{3, 16, 5},{3, 16, 6},{3, 16, 7},{3, 16, 8},{3, 16, 9},{3, 16, 10},{3, 16, 11},{3, 16, 12},{3, 16, 13},{3, 16, 14},{3, 16, 15},{3, 16, 16},{3, 16, 17},{3, 16, 18},{3, 16, 19},{3, 16, 20},{3, 16, 21},{3, 16, 22},{3, 16, 23},{3, 16, 24},{3, 16, 25},{3, 16, 26},{3, 16, 27},{3, 16, 28},{3, 16, 29},{3, 16, 30},{3, 16, 31},{3, 16, 32},{3, 16, 33},{3, 16, 34},{3, 16, 35},{3, 16, 36},{3, 16, 37},{3, 16, 38},{3, 16, 39},{3, 16, 40},{3, 16, 41},{3, 16, 42},{3, 16, 43},{3, 16, 44},{3, 16, 45},{3, 16, 46},{3, 16, 47},{3, 16, 48},{3, 16, 49},{3, 16, 50},{3, 16, 51},{3, 16, 52},{3, 16, 53},{3, 16, 54},{3, 16, 55},{3, 16, 56},{3, 16, 57},{3, 16, 58},{3, 16, 59},{3, 16, 60},{3, 16, 61},{3, 16, 62},{3, 16, 63},{3, 16, 64},{3, 16, 65},{3, 16, 66},{3, 16, 67},{3, 16, 68},{3, 16, 69},{3, 16, 70},{3, 16, 71},{3, 16, 72},{3, 16, 73},{3, 16, 74},{3, 16, 75},{3, 16, 76},{3, 16, 77},{3, 16, 78},{3, 16, 79},{3, 16, 80},{3, 16, 81},{3, 16, 82},{3, 16, 83},{3, 16, 84},{3, 16, 85},{3, 16, 86},{3, 16, 87},{3, 16, 88},{3, 16, 89},{3, 16, 90},{3, 16, 91},{3, 16, 92},{3, 16, 93},{3, 16, 94},{3, 16, 95},{3, 16, 96},{3, 16, 97},{3, 16, 98},{3, 16, 99},{3, 17, 0},{3, 17, 1},{3, 17, 2},{3, 17, 3},{3, 18, 0},{3, 18, 1},{3, 18, 2},{3, 18, 3},{3, 19, 0},{3, 19, 1},{3, 20, 0},{3, 20, 1},{3, 20, 2},{3, 20, 3},{3, 20, 4},{3, 20, 5},{3, 20, 6},{3, 20, 7},{3, 21, 0},{3, 21, 1},{3, 21, 2},{3, 21, 3},{3, 21, 4},{3, 21, 5},{3, 21, 6},{3, 21, 7},{3, 22, 0},{3, 22, 1},{3, 22, 2},{3, 22, 3},{3, 22, 4},{3, 22, 5},{3, 22, 6},{3, 22, 7},{3, 23, 0},{3, 23, 1},{3, 24, 0},{3, 24, 1},{3, 24, 2},{3, 24, 3},{3, 25, 0},{3, 25, 1},{3, 25, 2},{3, 25, 3},{3, 25, 4},{3, 25, 5},{3, 25, 6},{3, 25, 7},{3, 25, 8},{3, 25, 9},{3, 25, 10},{3, 25, 11},{3, 25, 12},{3, 25, 13},{3, 25, 14},{3, 25, 15},{3, 25, 16},{3, 25, 17},{3, 25, 18},{3, 25, 19},{3, 25, 20},{3, 25, 21},{3, 25, 22},{3, 25, 23},{3, 25, 24},{3, 25, 25},{3, 25, 26},{3, 25, 27},{3, 25, 28},{3, 25, 29},{3, 25, 30},{3, 25, 31},{3, 26, 0},{3, 26, 1},{3, 26, 2},{3, 26, 3},{3, 27, 0},{3, 27, 1},{3, 27, 2},{3, 27, 3},{3, 28, 0},{3, 28, 1},{3, 28, 2},{3, 28, 3},{3, 28, 4},{3, 28, 5},{3, 28, 6},{3, 28, 7},{3, 29, 0},{3, 29, 1},{3, 29, 2},{3, 29, 3},{3, 29, 4},{3, 29, 5},{3, 29, 6},{3, 29, 7},{3, 30, 0},{3, 30, 1},{3, 31, 0},{3, 31, 1},{3, 32, 0},{3, 33, 0},{3, 33, 1},{3, 33, 2},{3, 33, 3},{3, 34, 0},{3, 35, 0},{3, 35, 1},{3, 35, 2},{3, 35, 3},{3, 35, 4},{3, 35, 5},{3, 35, 6},{3, 35, 7},{3, 36, 0},{3, 37, 0},{3, 37, 1},{3, 38, 0},{3, 38, 1},{3, 38, 2},{3, 38, 3},{3, 39, 0},{3, 39, 1},{3, 39, 2},{3, 39, 3},{3, 39, 4},{3, 39, 5},{3, 39, 6},{3, 39, 7},{4, 0, 0},{4, 0, 1},{4, 1, 0},{4, 1, 1},{4, 1, 2},{4, 1, 3},{4, 1, 4},{4, 1, 5},{4, 1, 6},{4, 1, 7},{4, 1, 8},{4, 1, 9},{4, 1, 10},{4, 1, 11},{4, 1, 12},{4, 1, 13},{4, 1, 14},{4, 1, 15},{4, 2, 0},{4, 2, 1},{4, 3, 0},{4, 3, 1},{4, 4, 0},{4, 4, 1},{4, 4, 2},{4, 4, 3},{4, 5, 0},{4, 5, 1},{4, 5, 2},{4, 5, 3},{4, 5, 4},{4, 5, 5},{4, 5, 6},{4, 5, 7},{4, 6, 0},{4, 6, 1},{4, 6, 2},{4, 6, 3},{4, 7, 0},{4, 7, 1},{4, 7, 2},{4, 7, 3},{4, 7, 4},{4, 7, 5},{4, 7, 6},{4, 7, 7},{4, 7, 8},{4, 7, 9},{4, 7, 10},{4, 7, 11},{4, 7, 12},{4, 7, 13},{4, 7, 14},{4, 7, 15},{4, 8, 0},{4, 9, 0},{4, 9, 1},{4, 9, 2},{4, 9, 3},{4, 10, 0},{4, 10, 1},{4, 10, 2},{4, 10, 3},{4, 11, 0},{4, 11, 1},{4, 11, 2},{4, 11, 3},{4, 11, 4},{4, 11, 5},{4, 11, 6},{4, 11, 7},{4, 12, 0},{4, 12, 1},{4, 13, 0},{4, 13, 1},{4, 13, 2},{4, 13, 3},{4, 14, 0},{4, 14, 1},{4, 14, 2},{4, 14, 3},{4, 14, 4},{4, 14, 5},{4, 14, 6},{4, 14, 7},{4, 15, 0},{4, 16, 0},{4, 16, 1},{4, 16, 2},{4, 16, 3},{4, 16, 4},{4, 16, 5},{4, 16, 6},{4, 16, 7},{4, 16, 8},{4, 16, 9},{4, 16, 10},{4, 16, 11},{4, 16, 12},{4, 16, 13},{4, 16, 14},{4, 16, 15},{4, 16, 16},{4, 16, 17},{4, 16, 18},{4, 16, 19},{4, 16, 20},{4, 16, 21},{4, 16, 22},{4, 16, 23},{4, 16, 24},{4, 16, 25},{4, 16, 26},{4, 16, 27},{4, 16, 28},{4, 16, 29},{4, 16, 30},{4, 16, 31},{4, 16, 32},{4, 16, 33},{4, 16, 34},{4, 16, 35},{4, 16, 36},{4, 16, 37},{4, 16, 38},{4, 16, 39},{4, 16, 40},{4, 16, 41},{4, 16, 42},{4, 16, 43},{4, 16, 44},{4, 16, 45},{4, 16, 46},{4, 16, 47},{4, 16, 48},{4, 16, 49},{4, 16, 50},{4, 16, 51},{4, 16, 52},{4, 16, 53},{4, 16, 54},{4, 16, 55},{4, 16, 56},{4, 16, 57},{4, 16, 58},{4, 16, 59},{4, 16, 60},{4, 16, 61},{4, 16, 62},{4, 16, 63},{4, 17, 0},{4, 17, 1},{4, 18, 0},{4, 18, 1},{4, 19, 0},{4, 19, 1},{4, 20, 0},{4, 20, 1},{4, 20, 2},{4, 20, 3},{4, 21, 0},{4, 21, 1},{4, 21, 2},{4, 21, 3},{4, 22, 0},{4, 22, 1},{4, 22, 2},{4, 22, 3},{4, 22, 4},{4, 22, 5},{4, 22, 6},{4, 22, 7},{4, 23, 0},{4, 23, 1},{4, 24, 0},{4, 24, 1},{4, 25, 0},{4, 25, 1},{4, 25, 2},{4, 25, 3},{4, 25, 4},{4, 25, 5},{4, 25, 6},{4, 25, 7},{4, 26, 0},{4, 27, 0},{4, 28, 0},{4, 28, 1},{4, 28, 2},{4, 28, 3},{4, 29, 0},{4, 29, 1},{4, 29, 2},{4, 29, 3},{4, 29, 4},{4, 29, 5},{4, 29, 6},{4, 29, 7},{4, 29, 8},{4, 29, 9},{4, 29, 10},{4, 29, 11},{4, 29, 12},{4, 29, 13},{4, 29, 14},{4, 29, 15},{4, 30, 0},{4, 31, 0},{4, 32, 0},{4, 32, 1},{4, 32, 2},{4, 32, 3},{4, 32, 4},{4, 32, 5},{4, 32, 6},{4, 32, 7},{4, 33, 0},{4, 33, 1},{4, 34, 0},{4, 35, 0},{4, 35, 1},{4, 35, 2},{4, 35, 3},{4, 35, 4},{4, 35, 5},{4, 35, 6},{4, 35, 7},{4, 35, 8},{4, 35, 9},{4, 35, 10},{4, 35, 11},{4, 35, 12},{4, 35, 13},{4, 35, 14},{4, 35, 15},{4, 36, 0},{4, 37, 0},{4, 37, 1},{4, 38, 0},{4, 38, 1},{4, 39, 0},{4, 39, 1},{5, 0, 0},{5, 0, 1},{5, 1, 0},{5, 2, 0},{5, 2, 1},{5, 3, 0},{5, 4, 0},{5, 5, 0},{5, 5, 1},{5, 5, 2},{5, 5, 3},{5, 5, 4},{5, 5, 5},{5, 5, 6},{5, 5, 7},{5, 6, 0},{5, 7, 0},{5, 8, 0},{5, 9, 0},{5, 9, 1},{5, 10, 0},{5, 11, 0},{5, 12, 0},{5, 13, 0},{5, 14, 0},{5, 14, 1},{5, 14, 2},{5, 14, 3},{5, 15, 0},{5, 16, 0},{5, 17, 0},{5, 18, 0},{5, 19, 0},{5, 19, 1},{5, 19, 2},{5, 19, 3},{5, 20, 0},{5, 21, 0},{5, 22, 0},{5, 23, 0},{5, 23, 1},{5, 24, 0},{5, 25, 0},{5, 26, 0},{5, 27, 0},{5, 28, 0},{5, 29, 0},{5, 29, 1},{5, 29, 2},{5, 29, 3},{5, 30, 0},{5, 31, 0},{5, 32, 0},{5, 32, 1},{5, 32, 2},{5, 32, 3},{5, 32, 4},{5, 32, 5},{5, 32, 6},{5, 32, 7},{5, 33, 0},{5, 34, 0},{5, 35, 0},{5, 35, 1},{5, 35, 2},{5, 35, 3},{5, 36, 0},{5, 37, 0},{5, 37, 1},{5, 37, 2},{5, 37, 3},{5, 38, 0},{5, 39, 0}};


// Kernel function to add the elements of two arrays
__global__ void v_set(unsigned int *arr, float val, int m)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index < m)
    arr[index] = val;
}

// Kernel function to add the elements of two arrays
__global__ void v_set(int *arr, float val, int m)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index < m)
    arr[index] = val;
}

// Kernel function to add the elements of two arrays
__global__ void v_set(float *arr, float val, int m)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index < m)
    arr[index] = val;
}


std::fstream& GotoLine(std::fstream& file, unsigned int num){
    file.seekg(std::ios::beg);
    for(int i=0; i < num - 1; ++i){
        file.ignore(std::numeric_limits<std::streamsize>::max(),'\n');
    }
    return file;
}

void printDesignPoint(float* designSpaceTensor, unsigned int dpIndex)
{
  unsigned int peCount = 37;
  unsigned int funcCount = 38;
  for(int funcTypeIdx = 0; funcTypeIdx<funcCount; funcTypeIdx++)
  {
    for(int peIndex = 0; peIndex<peCount; peIndex++)
    {
      printf("%d ", (int)(designSpaceTensor[dpIndex*(peCount*funcCount) + funcTypeIdx*peCount + peIndex]));
    }
    printf("\n");
  }
}

float* loadDSTensor(unsigned int& designPointsCount, unsigned int& peCount, unsigned int& funcCount)
{

  vector<vector<vector<unsigned int>>> designPoints;

  for(int i = 0; i<1047; i++)
  {
    string designPointFileString = "data/arch";
    designPointFileString += std::to_string(fileInfoArray[i][0]);
    designPointFileString += "_app";
    designPointFileString += std::to_string(fileInfoArray[i][1]);
    designPointFileString += "_bind";
    designPointFileString += std::to_string(fileInfoArray[i][2]);
    designPointFileString += ".txt";
    std::fstream designPointFile(designPointFileString);
    GotoLine(designPointFile, 6);
    string line;
    vector<vector<unsigned int>> designPoint;
    while(std::getline(designPointFile, line))
    {
      std::istringstream ss(line);
      unsigned int inputSize;
      vector<unsigned int> functionInputs;
      while(ss >> inputSize)
      {
        functionInputs.push_back(inputSize);
      }
      designPoint.push_back(functionInputs);
    }
    designPoints.push_back(designPoint);
  }

  vector<vector<unsigned int>> archVector;
  vector<unsigned int> allocatedPEs;
  for(int i = 0; i<6; i++)
  {
    string archFileString = "data/arch";
    archFileString += std::to_string(i);
    archFileString += ".txt";
    std::fstream archFile(archFileString);
    
    GotoLine(archFile, 6);
    string line;
    std::getline(archFile, line);
    unsigned int allocatedPe;
    std::istringstream ss(line);
    while(ss >> allocatedPe)
    {
      allocatedPEs.push_back(allocatedPe);
    }
    archVector.push_back(allocatedPEs);
    allocatedPEs.clear();
  }

  designPointsCount = 1047;
  peCount = 37;
  funcCount = 38;

  float *designSpaceTensor;
  unsigned int designSpaceSize = designPointsCount*peCount*funcCount;

  hipMallocManaged(&designSpaceTensor, designSpaceSize*sizeof(float));
  
  unsigned int threadCount = 1024;
  unsigned int blockCount = (designSpaceSize+(threadCount-1))/threadCount;

  v_set KERNEL_ARG2(blockCount,threadCount)(designSpaceTensor, 0, designSpaceSize);
  hipDeviceSynchronize();

  for(auto dpIndex = 0; dpIndex < designPoints.size(); dpIndex++)
  {
    for(auto funcTypeIndex = 0; funcTypeIndex < designPoints[dpIndex].size(); funcTypeIndex++)
    {
      for(auto peIndex = 0; peIndex < designPoints[dpIndex][funcTypeIndex].size(); peIndex++)
      {
        unsigned int peInputSize = designPoints[dpIndex][funcTypeIndex][peIndex];
        unsigned int tensorPeIndex = archVector[fileInfoArray[dpIndex][0]][peIndex];
        designSpaceTensor[dpIndex*(peCount*funcCount) + funcTypeIndex*peCount + tensorPeIndex] = peInputSize;
      } 
    }
  }

  return designSpaceTensor;
}

float* loadPerfTable(unsigned int& coefficientCount, unsigned int peCount, unsigned int funcCount)
{
  float c0[1406] = {0, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, 15, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, 25, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, 25, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, 45, 45, 50, 50, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 835, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 15, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 15, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 75, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, 45, 45, 50, 50, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, 50, 50, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 25, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 105, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 25, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, 45, 45, 50, 50, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 12.5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 155, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 15, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 12.5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 140, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 31.5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 345, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 25, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 225, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 200, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 105, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 50, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 25, 25, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 105, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 570, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, 50, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 45, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 120, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 50, -1, -1, -1, -1, -1, -1, 45 };
  float c1[1406] = {0.5, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 2, -1, 20, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0.5, -1, -1, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 3, -1, -1, -1, 30, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 5, -1, -1, -1, -1, 30, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0.5, -1, -1, -1, -1, -1, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 40, -1, -1, -1, -1, -1, -1, 50, 50, 55, 55, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 314, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 840, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 20, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 20, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 39, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 80, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 40, -1, -1, -1, -1, -1, -1, 50, 50, 55, 55, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 112, -1, -1, -1, -1, -1, -1, -1, -1, 55, 55, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 22, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 30, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 110, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 22, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 30, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 40, -1, -1, -1, -1, -1, -1, 50, 50, 55, 55, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 28, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 13.75, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 570, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 160, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 3.5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 20, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 28, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 13.75, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 16.5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 145, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 35, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 355, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 26, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 30, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0.5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 671.5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 230, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 20.5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 205, -1, -1, -1, -1, -1, -1, -1, -1, -1, 5.5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 106.25, -1, -1, -1, -1, -1, -1, -1, -1, 224, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 55, -1, -1, -1, -1, -1, -1, -1, 0.5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 5, -1, -1, -1, -1, -1, -1, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 30, 30, -1, -1, -1, -1, 31, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 110, -1, -1, -1, -1, 57.5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 575, -1, -1, -1, 112, -1, -1, -1, -1, -1, -1, -1, -1, -1, 55, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 50, -1, -1, 31.5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 125, -1, 80, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 55, -1, -1, -1, -1, -1, -1, 50};

  float* perfTable;
  coefficientCount = 2;
  unsigned int perfTableSize = peCount*funcCount;
  hipMallocManaged(&perfTable, perfTableSize*sizeof(float));

  unsigned int threadCount = 1024;
  unsigned int blockCount = (perfTableSize+(threadCount-1))/threadCount;

  v_set KERNEL_ARG2(blockCount,threadCount)(perfTable, 0, perfTableSize);
  hipDeviceSynchronize();

  for(int i = 0; i<peCount*funcCount; i++)
  {
    perfTable[i] = c0[i];
  }

  for(int i = 0; i<peCount*funcCount; i++)
  {
    perfTable[peCount*funcCount + i] = c1[i];
  }

  return perfTable;
}

// __global__ void transposeCoalesced(float *odata, const float *idata)
// {
//   __shared__ float tile[TILE_DIM][TILE_DIM];

//   int x = blockIdx.x * TILE_DIM + threadIdx.x;
//   int y = blockIdx.y * TILE_DIM + threadIdx.y;
//   int width = gridDim.x * TILE_DIM;

//   for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
//      tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*width + x];

//   __syncthreads();

//   x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
//   y = blockIdx.x * TILE_DIM + threadIdx.y;

//   for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
//      odata[(y+j)*width + x] = tile[threadIdx.x][threadIdx.y + j];
// }


// Kernel function to add the elements of two arrays
__global__ void t_mult(float *designSpaceTensor, float *perfTable, float* latencyTensor, int designPointsCount,int peCount,int funcCount)
{
  int designPointSize = peCount*funcCount;
  int thrdDesignPointIndex = blockIdx.x * blockDim.x + threadIdx.x;
  int perfTableC1Offset = designPointSize;
  int globalIndex = blockIdx.y*designPointSize + thrdDesignPointIndex;
  int stride = peCount;
  if(thrdDesignPointIndex < peCount)
  {
    for (int dpIdx = globalIndex, perfIdx = thrdDesignPointIndex; perfIdx < designPointSize; dpIdx += stride, perfIdx += stride)
    {
        latencyTensor[dpIdx] = perfTable[perfIdx] + designSpaceTensor[dpIdx]*perfTable[perfIdx + perfTableC1Offset];
    }
  }
}

__global__ void t_vreduce_sum(float* latencyTensor, float* aggregateLatencyMatrix, int designPointsCount,int peCount,int funcCount)
{
  int designPointSize = peCount*funcCount;
  int thrdDesignPointIndex = blockIdx.x * blockDim.x + threadIdx.x;
  int globalIndex = blockIdx.y*designPointSize + thrdDesignPointIndex;
  int stride = peCount;
  if(thrdDesignPointIndex < peCount)
  {
    float sum = 0;
    for (int dpIdx = globalIndex, perfIdx = thrdDesignPointIndex; perfIdx < designPointSize; dpIdx += stride, perfIdx += stride)
    {
        sum += latencyTensor[dpIdx];
    }
    //naive transpose horrible performance due no coalescing
    aggregateLatencyMatrix[thrdDesignPointIndex*designPointsCount + blockIdx.y] = sum;
  }
}

__global__ void m_max(float* aggregateLatencyMatrix, float* maxLatencyVector, int designPointsCount,int peCount)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = designPointsCount;
  if(index < designPointsCount)
  {
    float max = 0;
    for (int i = index; i < designPointsCount*peCount; i += stride)
    {
      float mVal = aggregateLatencyMatrix[i];

      if(max < mVal)
      {
        max = mVal;
      }

      __syncthreads();
    }

    maxLatencyVector[index] = max;
  }
}

float* createAndSetCudaManagedMemory(unsigned int size)
{
  float* mem; 
  hipMallocManaged(&mem, size*sizeof(float));

  unsigned int threadCount = 1024;
  unsigned int blockCount = (size+(threadCount-1))/threadCount;

  v_set KERNEL_ARG2(blockCount,threadCount)(mem, 0, size);
  hipDeviceSynchronize();

  return mem;
}

void validateLatencyTensor(float* latencyTensor, float* designSpaceTensor, float* perfTable, unsigned int designPointsCount, unsigned int funcCount, unsigned int peCount)
{
  unsigned int designPointSize = peCount*funcCount;
  for(int dpIdx = 0; dpIdx < designPointsCount; dpIdx++)
  {
    for(int funcTypeIdx = 0; funcTypeIdx<funcCount; funcTypeIdx++)
    {
      for(int peIndex = 0; peIndex<peCount; peIndex++)
      {
        unsigned int gidx = dpIdx*designPointSize + funcTypeIdx*peCount + peIndex;
        unsigned int pidx = funcTypeIdx*peCount + peIndex;
        float expectedVal = designSpaceTensor[gidx]*perfTable[pidx+designPointSize] + perfTable[pidx];
        assert((latencyTensor[gidx] - expectedVal) < 0.001);
      }
    }
  }
}


void validateAggregateLatencyMatrix(float* latencyTensor, float* aggregateLatencyMatrix, unsigned int designPointsCount, unsigned int funcCount, unsigned int peCount)
{
  unsigned int designPointSize = peCount*funcCount;
  for(int dpIdx = 0; dpIdx < designPointsCount; dpIdx++)
  {
    for(int peIndex = 0; peIndex<peCount; peIndex++)
    {
      float sum = 0;
      for(int funcTypeIdx = 0; funcTypeIdx<funcCount; funcTypeIdx++)
      {
        unsigned int gidx = dpIdx*designPointSize + funcTypeIdx*peCount + peIndex;
        sum += latencyTensor[gidx];
      }
      assert((sum - aggregateLatencyMatrix[peIndex*designPointsCount + dpIdx]) < 0.001);
    }
  }
}

void validateMaxLatencyVector(float* aggregateLatencyMatrix, float* latencyVector, unsigned int designPointsCount, unsigned int peCount)
{
  for(int dpIdx = 0; dpIdx < designPointsCount; dpIdx++)
  {
    float max = 0;
    for(int peIndex = 0; peIndex<peCount; peIndex++)
    {
      float val = aggregateLatencyMatrix[peIndex*designPointsCount + peIndex];
      if(val > max)
      {
        max = val;
      }
    }
    assert((max - latencyVector[dpIdx]) < 0.001);
  }
}

int main(void)
{
  
  unsigned int designPointsCount;
  unsigned int peCount;
  unsigned int funcCount;
  unsigned int coefficientCount;

  float* designSpaceTensor = loadDSTensor(designPointsCount, peCount, funcCount);
  float* perfTable = loadPerfTable(coefficientCount, peCount, funcCount);

  unsigned int designSpaceSize = designPointsCount*peCount*funcCount;

  auto start = high_resolution_clock::now(); 

  float* latencyTensor = createAndSetCudaManagedMemory(designSpaceSize);
  float* aggregateLatencyMatrix = createAndSetCudaManagedMemory(designPointsCount*peCount);
  float* maxLatencyVector = createAndSetCudaManagedMemory(designPointsCount);
  
  dim3 dimGrid;
  unsigned int threadCount = 32;
  dimGrid.x = (peCount+(threadCount-1))/threadCount;
  dimGrid.y = designPointsCount;

  t_mult KERNEL_ARG2(dimGrid,threadCount)(designSpaceTensor,perfTable,latencyTensor,designPointsCount,peCount,funcCount);
  hipDeviceSynchronize();
  
  #ifdef DEBUG
  validateLatencyTensor(latencyTensor, designSpaceTensor, perfTable, designPointsCount, funcCount, peCount);
  #endif

  t_vreduce_sum KERNEL_ARG2(dimGrid,threadCount)(latencyTensor, aggregateLatencyMatrix, designPointsCount, peCount, funcCount);
  hipDeviceSynchronize();
  
  #ifdef DEBUG
  validateAggregateLatencyMatrix(latencyTensor, aggregateLatencyMatrix, designPointsCount, funcCount, peCount);
  #endif 

  threadCount = 32;
  unsigned int blockCount = (designPointsCount+(threadCount-1))/threadCount;

  m_max KERNEL_ARG2(blockCount,threadCount)(aggregateLatencyMatrix, maxLatencyVector, designPointsCount, peCount);
  hipDeviceSynchronize();

  #ifdef DEBUG
  validateMaxLatencyVector(aggregateLatencyMatrix,maxLatencyVector, designPointsCount, peCount);
  #endif
  
  auto stop = high_resolution_clock::now(); 

  auto duration = duration_cast<microseconds>(stop - start); 

  printf("Time to Evaluate:  %lu us \n", duration.count());

  for(int i = 0; i<1047; i++)
  {
    printf("arch%d_app%d_bind%d maxLatency:%.02f\n", fileInfoArray[i][0], fileInfoArray[i][1], fileInfoArray[i][2], maxLatencyVector[i]);
  }

  return 0;
}
