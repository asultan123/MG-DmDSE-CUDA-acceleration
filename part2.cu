#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <sys/time.h>
#include "cudaDmy.cuh"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>
#include <fstream>
#include <map>
#include <set> 
#include <string>
#include <vector>
#include <iterator>
#include <algorithm>
#include <bits/stdc++.h> 
#include <assert.h>
#include <limits>
#include <string>
#include <sstream>

using std::ifstream;
using std::cout;
using std::endl;
using std::map;
using std::set;
using std::pair;
using std::make_pair;
using std::string;
using std::vector;
using std::advance;
using std::sort;
using std::accumulate;
using std::max_element;

struct pairCompareSecondDescending {
    bool operator() (const pair<int,int> &lhs, const pair<int,int> &rhs) const{
         return (lhs.second > rhs.second);
    }
};

struct timeval t1, t2;

__global__ void m_find_coauthors(const unsigned int *arr, int m, int n, unsigned int* coAuthorCount)
{

  __shared__ unsigned int smem[2];
  unsigned int *bSum = &smem[0];
  unsigned int *findTarget = &smem[1];
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = m;

  // thread 0 in block sets smem bSum to 0
  if(threadIdx.x == 0)
  {
    *bSum = 0;
  }

  __syncthreads();

  for(int i = 0; i<m ; i++)
  {
    for(int j = 0; j<n; j++)
    {
      // thread 0 in block sets findTarget for all block
      // find target will be the same for all blocks
      if(threadIdx.x == 0)
      {
        *findTarget = arr[j*m+i];
        // printf("THREAD %d set findTarget to %d!\n", index, *findTarget);
      }

      // sync due to divergence
      __syncthreads();

      // since num threads are guaranteed to be >= m
      // the thread that has an index equal to current 
      // author being explored is dropped 

        // all threads traverse down in order to 
        // look if target is in their "lane"
        // if target is present block bSum is incremented
        // sync called due to possible divergence
      for (int thrd = index; thrd < m*n; thrd += stride)
      {
          if(arr[thrd] == *findTarget && index != i)
          {
            atomicAdd(bSum, 1);
          }
          __syncthreads();
      }
    }
  }

  // aggregate bSums to coAuthorCount
  if(threadIdx.x == 0)
  {
    atomicAdd(coAuthorCount, *bSum);
  }

}

__global__ void v_bucket(const unsigned int *arr, unsigned int *gbucket, int arrSize, int sbucketSize)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  
  // Dynamically allocated shared memory array 
  extern __shared__ unsigned int sbuckets[];

  // This loop forces blocks to prep their own shared memory
  // Shared memory is not shared accross blocks, it's shared across
  // threads. As a result indexing is controlled by threads
  // Interleaved access used here for thread coalescing. Stride is 
  // number of threads in a block because only 1 block accesses the 
  // shared memory so it's equivelent to blockDim.x * gridDim.x 
  // where gridDim.x is 1. 
  for (int i = threadIdx.x; i < sbucketSize; i += blockDim.x)
  {
    sbuckets[i] = 0;
  }

  // synchronize idle threads that skipped loop with threads that
  // went through loop. 
  __syncthreads();


  int stride = blockDim.x * gridDim.x;

  //if index exceeds size of sbucket (block with extra threads)
  //condition for loop will fail and those threads will be idle
  //stride prevents interleaved threads from one block overlapping
  //with threads from another block. Kernel is generally always 
  //launched with blocks and threads that exceed the size of the 
  //array being bucketed but if less the += stride fixes it. 
  for(int i = index; i<arrSize; i+=stride)
  {
    atomicAdd(&(sbuckets[arr[i]]), 1);
  }

  // synchronize idle threads that skipped loop with threads that
  // went through loop. 
  __syncthreads();

  // each block goes through own shared memory, and aggregates
  // value in index i of own shared memory into index i of gbucket
  // gbucket and shared memory bucket sizes are identical. Access
  // of shared memory is by block so indexing of loop is similair 
  // to setting shared memory to zero. Middle loop access involved 
  // more blocks accessing input array so indexing was different.
  // If blockDim.x*threadIdx.x > sbucketSize then some threads will
  // be idle. Hence, there should be a need dfor __syncthreads but
  // since this is the last operation in the kernel we don't need 
  // to syncthreads. Atomic add still used because multiple threads in blocks 
  // will access the same index in gbucket. This could potentially be
  // avoided if each block accessed seperate parts of their own shared
  // memory at different offsets from each other (thus eliminating the
  // need for atomicAdd because no two blocks would access the same indexes)
  // however, that's too hard and the current implementation is fast enough

  for(int i = threadIdx.x; i < sbucketSize; i += blockDim.x)
  {
    atomicAdd(&(gbucket[i]), sbuckets[i]);
  }

}

// Kernel function to add the elements of two arrays
__global__ void v_accumulate(unsigned int *arr, unsigned int *acc, int m, int n)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = m;
  if(index < m)
  {
    for (int i = index; i < m*n; i += stride)
    {
        acc[index] += (arr[i] >= 1)? 1 : 0;
    }
  }
}

// Kernel function to add the elements of two arrays
__global__ void v_set(unsigned int *arr, float val, int m)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index < m)
    arr[index] = val;
}

// Kernel function to add the elements of two arrays
__global__ void v_set(int *arr, float val, int m)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index < m)
    arr[index] = val;
}

// Kernel function to add the elements of two arrays
__global__ void v_set(float *arr, float val, int m)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index < m)
    arr[index] = val;
}

map<unsigned int, unsigned int> coAuthorCountHistogram(unsigned int *coAuthorCounts, int m, int n, unsigned int maxCoAuthorCount)
{
  unsigned int threadCount = 1024;
  unsigned int blockCount = (maxCoAuthorCount+(threadCount-1))/threadCount;

  unsigned int* gbuckets = NULL; 
  hipMallocManaged(&gbuckets, (maxCoAuthorCount)*sizeof(unsigned int));

  v_set KERNEL_ARG2(blockCount,threadCount)(gbuckets, 0, maxCoAuthorCount);
  hipDeviceSynchronize();

  threadCount = 1024;
  blockCount = (m+(threadCount-1))/threadCount;

  v_bucket KERNEL_ARG3(blockCount,threadCount, maxCoAuthorCount*sizeof(unsigned int)) (coAuthorCounts, gbuckets, m, maxCoAuthorCount);
  hipDeviceSynchronize();

  map<unsigned int, unsigned int> histogram;

  for(int i = 0; i<maxCoAuthorCount; i++)
  {
    if(gbuckets[i] != 0)
    {
      histogram[i] = gbuckets[i];
    }
  }
  
  hipFree(gbuckets);

  return histogram;
}

map<int, set< int>> loadcoAuthors()
{
  ifstream infile("coauthors.txt");
  map<int, set<int>> graph;
  vector<int> node1arr, node2arr;
  int node1, node2; 
  while(infile >> node1 >> node2)
  {
      node1arr.push_back(node1);
      node2arr.push_back(node2);
  }
  for(int i = 0; i < node1arr.size(); i++)
  {
      graph[node1arr[i]].insert(node2arr[i]);
      graph[node2arr[i]].insert(node1arr[i]);
  }
  return graph;
}

unsigned int* flattenGraph(map<int, set<int>> graph, int& m, int& n)
{
  m = graph.size()+1;
  unsigned int* arr;

  n = 0;
  for(auto node : graph)
  {
    if(node.second.size() > n)
    {
      n = node.second.size();
    }
  }

  hipMallocManaged(&arr, ((m)*n)*sizeof(unsigned int));

  unsigned int threadCount = 1024;
  unsigned int blockCount = ((m*n)+(threadCount-1))/threadCount;
  v_set KERNEL_ARG2(blockCount,threadCount)(arr, 0, m*n);
  hipDeviceSynchronize();

  for(int i = 0; i<m; i++)
  {
    if(graph.find(i) != graph.end())
    {
      vector<unsigned int> coAuthors(graph[i].begin(), graph[i].end());
      for(int j = 0; j<coAuthors.size(); j++)
      {
        arr[j*m+i] = coAuthors[j];
      }
    }
  }
  return arr;
}

set<pair<int,int>, pairCompareSecondDescending> rankAuthorByCoAuthorCount(unsigned int* arr, unsigned int **coAuthorSums, int m, int n)
{
  set<pair<int,int>, pairCompareSecondDescending> rankings;
  hipMallocManaged(coAuthorSums, (m)*sizeof(unsigned int));

  unsigned int threadCount = 1024;
  unsigned int blockCount = (m+(threadCount-1))/threadCount;

  v_set KERNEL_ARG2(blockCount,threadCount)(*coAuthorSums, 0, m);
  hipDeviceSynchronize();

  v_accumulate KERNEL_ARG2(blockCount,threadCount)(arr, *coAuthorSums, m, n);
  hipDeviceSynchronize();

  for(int i = 0; i<m; i++)
  {
    rankings.insert(make_pair(i, (*coAuthorSums)[i]));
  }

  return rankings;
}

void printRankings(set<pair<int,int>, pairCompareSecondDescending> rankings)
{
  for(auto it = rankings.begin(); it != rankings.end(); it++)
  {
    printf("Author[%d] coauthorCount = %d\n", it->first, it->second);
  }
}

unsigned int getAuthorThatShareCoAuthorCount(const unsigned int *arr, int m, int n)
{
  unsigned int* count ;
  hipMallocManaged(&count, sizeof(unsigned int));
  
  unsigned int threadCount = 1024;
  unsigned int blockCount = (m+(threadCount-1))/threadCount;
  // unsigned int blockCount = 512;
  
  printf("Thread Count per block %d\n", threadCount);
  printf("blockCount %d\n", blockCount);
  printf("Total Threads %d\n", blockCount*threadCount);

  m_find_coauthors KERNEL_ARG2(blockCount, threadCount)(arr, m, n, count);
  hipDeviceSynchronize();

  unsigned int returnable = *count;

  hipFree(count);

  return returnable;
}

std::fstream& GotoLine(std::fstream& file, unsigned int num){
    file.seekg(std::ios::beg);
    for(int i=0; i < num - 1; ++i){
        file.ignore(std::numeric_limits<std::streamsize>::max(),'\n');
    }
    return file;
}

void printDesignPoint(float* designSpaceTensor, unsigned int dpIndex)
{
  unsigned int peCount = 37;
  unsigned int funcCount = 38;
  for(int funcTypeIdx = 0; funcTypeIdx<funcCount; funcTypeIdx++)
  {
    for(int peIndex = 0; peIndex<peCount; peIndex++)
    {
      printf("%d ", (int)(designSpaceTensor[dpIndex*(peCount*funcCount) + funcTypeIdx*peCount + peIndex]));
    }
    printf("\n");
  }
}

float* loadDSTensor(unsigned int& designPointsCount, unsigned int& peCount, unsigned int& funcCount)
{
  unsigned int fileInfoArray[1047][3] = {{0, 0, 0},{0, 1, 0},{0, 2, 0},{0, 3, 0},{0, 4, 0},{0, 5, 0},{0, 6, 0},{0, 7, 0},{0, 8, 0},{0, 9, 0},{0, 10, 0},{0, 11, 0},{0, 12, 0},{0, 13, 0},{0, 14, 0},{0, 15, 0},{0, 16, 0},{0, 17, 0},{0, 18, 0},{0, 19, 0},{0, 20, 0},{0, 21, 0},{0, 22, 0},{0, 23, 0},{0, 24, 0},{0, 25, 0},{0, 26, 0},{0, 27, 0},{0, 28, 0},{0, 29, 0},{0, 30, 0},{0, 31, 0},{0, 32, 0},{0, 33, 0},{0, 34, 0},{0, 35, 0},{0, 36, 0},{0, 37, 0},{0, 38, 0},{0, 39, 0},{1, 0, 0},{1, 0, 1},{1, 1, 0},{1, 2, 0},{1, 2, 1},{1, 3, 0},{1, 4, 0},{1, 4, 1},{1, 5, 0},{1, 5, 1},{1, 5, 2},{1, 5, 3},{1, 6, 0},{1, 6, 1},{1, 7, 0},{1, 8, 0},{1, 9, 0},{1, 9, 1},{1, 10, 0},{1, 10, 1},{1, 11, 0},{1, 12, 0},{1, 13, 0},{1, 13, 1},{1, 14, 0},{1, 14, 1},{1, 14, 2},{1, 14, 3},{1, 15, 0},{1, 16, 0},{1, 17, 0},{1, 18, 0},{1, 19, 0},{1, 19, 1},{1, 20, 0},{1, 21, 0},{1, 22, 0},{1, 23, 0},{1, 24, 0},{1, 25, 0},{1, 25, 1},{1, 26, 0},{1, 27, 0},{1, 27, 1},{1, 28, 0},{1, 28, 1},{1, 29, 0},{1, 29, 1},{1, 29, 2},{1, 29, 3},{1, 30, 0},{1, 31, 0},{1, 32, 0},{1, 32, 1},{1, 32, 2},{1, 32, 3},{1, 32, 4},{1, 32, 5},{1, 32, 6},{1, 32, 7},{1, 33, 0},{1, 34, 0},{1, 35, 0},{1, 35, 1},{1, 35, 2},{1, 35, 3},{1, 36, 0},{1, 37, 0},{1, 37, 1},{1, 38, 0},{1, 38, 1},{1, 39, 0},{1, 39, 1},{2, 0, 0},{2, 1, 0},{2, 1, 1},{2, 1, 2},{2, 1, 3},{2, 1, 4},{2, 1, 5},{2, 1, 6},{2, 1, 7},{2, 1, 8},{2, 1, 9},{2, 1, 10},{2, 1, 11},{2, 1, 12},{2, 1, 13},{2, 1, 14},{2, 1, 15},{2, 2, 0},{2, 3, 0},{2, 3, 1},{2, 3, 2},{2, 3, 3},{2, 4, 0},{2, 4, 1},{2, 4, 2},{2, 4, 3},{2, 4, 4},{2, 4, 5},{2, 4, 6},{2, 4, 7},{2, 5, 0},{2, 5, 1},{2, 6, 0},{2, 6, 1},{2, 6, 2},{2, 6, 3},{2, 6, 4},{2, 6, 5},{2, 6, 6},{2, 6, 7},{2, 7, 0},{2, 7, 1},{2, 7, 2},{2, 7, 3},{2, 7, 4},{2, 7, 5},{2, 7, 6},{2, 7, 7},{2, 7, 8},{2, 7, 9},{2, 7, 10},{2, 7, 11},{2, 7, 12},{2, 7, 13},{2, 7, 14},{2, 7, 15},{2, 8, 0},{2, 9, 0},{2, 9, 1},{2, 10, 0},{2, 10, 1},{2, 10, 2},{2, 10, 3},{2, 10, 4},{2, 10, 5},{2, 10, 6},{2, 10, 7},{2, 11, 0},{2, 11, 1},{2, 11, 2},{2, 11, 3},{2, 11, 4},{2, 11, 5},{2, 11, 6},{2, 11, 7},{2, 12, 0},{2, 12, 1},{2, 13, 0},{2, 13, 1},{2, 13, 2},{2, 13, 3},{2, 13, 4},{2, 13, 5},{2, 13, 6},{2, 13, 7},{2, 14, 0},{2, 14, 1},{2, 15, 0},{2, 16, 0},{2, 16, 1},{2, 16, 2},{2, 16, 3},{2, 16, 4},{2, 16, 5},{2, 16, 6},{2, 16, 7},{2, 16, 8},{2, 16, 9},{2, 16, 10},{2, 16, 11},{2, 16, 12},{2, 16, 13},{2, 16, 14},{2, 16, 15},{2, 16, 16},{2, 16, 17},{2, 16, 18},{2, 16, 19},{2, 16, 20},{2, 16, 21},{2, 16, 22},{2, 16, 23},{2, 16, 24},{2, 16, 25},{2, 16, 26},{2, 16, 27},{2, 16, 28},{2, 16, 29},{2, 16, 30},{2, 16, 31},{2, 16, 32},{2, 16, 33},{2, 16, 34},{2, 16, 35},{2, 16, 36},{2, 16, 37},{2, 16, 38},{2, 16, 39},{2, 16, 40},{2, 16, 41},{2, 16, 42},{2, 16, 43},{2, 16, 44},{2, 16, 45},{2, 16, 46},{2, 16, 47},{2, 16, 48},{2, 16, 49},{2, 16, 50},{2, 16, 51},{2, 16, 52},{2, 16, 53},{2, 16, 54},{2, 16, 55},{2, 16, 56},{2, 16, 57},{2, 16, 58},{2, 16, 59},{2, 16, 60},{2, 16, 61},{2, 16, 62},{2, 16, 63},{2, 17, 0},{2, 17, 1},{2, 18, 0},{2, 18, 1},{2, 19, 0},{2, 20, 0},{2, 20, 1},{2, 20, 2},{2, 20, 3},{2, 21, 0},{2, 21, 1},{2, 21, 2},{2, 21, 3},{2, 22, 0},{2, 22, 1},{2, 22, 2},{2, 22, 3},{2, 22, 4},{2, 22, 5},{2, 22, 6},{2, 22, 7},{2, 23, 0},{2, 23, 1},{2, 24, 0},{2, 24, 1},{2, 25, 0},{2, 25, 1},{2, 25, 2},{2, 25, 3},{2, 25, 4},{2, 25, 5},{2, 25, 6},{2, 25, 7},{2, 25, 8},{2, 25, 9},{2, 25, 10},{2, 25, 11},{2, 25, 12},{2, 25, 13},{2, 25, 14},{2, 25, 15},{2, 26, 0},{2, 27, 0},{2, 27, 1},{2, 28, 0},{2, 28, 1},{2, 28, 2},{2, 28, 3},{2, 28, 4},{2, 28, 5},{2, 28, 6},{2, 28, 7},{2, 29, 0},{2, 29, 1},{2, 29, 2},{2, 29, 3},{2, 30, 0},{2, 31, 0},{2, 32, 0},{2, 33, 0},{2, 33, 1},{2, 34, 0},{2, 35, 0},{2, 35, 1},{2, 35, 2},{2, 35, 3},{2, 36, 0},{2, 37, 0},{2, 38, 0},{2, 38, 1},{2, 38, 2},{2, 38, 3},{2, 38, 4},{2, 38, 5},{2, 38, 6},{2, 38, 7},{2, 39, 0},{2, 39, 1},{2, 39, 2},{2, 39, 3},{2, 39, 4},{2, 39, 5},{2, 39, 6},{2, 39, 7},{3, 0, 0},{3, 0, 1},{3, 1, 0},{3, 1, 1},{3, 1, 2},{3, 1, 3},{3, 1, 4},{3, 1, 5},{3, 1, 6},{3, 1, 7},{3, 1, 8},{3, 1, 9},{3, 1, 10},{3, 1, 11},{3, 1, 12},{3, 1, 13},{3, 1, 14},{3, 1, 15},{3, 2, 0},{3, 2, 1},{3, 3, 0},{3, 3, 1},{3, 3, 2},{3, 3, 3},{3, 4, 0},{3, 4, 1},{3, 4, 2},{3, 4, 3},{3, 4, 4},{3, 4, 5},{3, 4, 6},{3, 4, 7},{3, 4, 8},{3, 4, 9},{3, 4, 10},{3, 4, 11},{3, 4, 12},{3, 4, 13},{3, 4, 14},{3, 4, 15},{3, 5, 0},{3, 5, 1},{3, 5, 2},{3, 5, 3},{3, 6, 0},{3, 6, 1},{3, 6, 2},{3, 6, 3},{3, 6, 4},{3, 6, 5},{3, 6, 6},{3, 6, 7},{3, 6, 8},{3, 6, 9},{3, 6, 10},{3, 6, 11},{3, 6, 12},{3, 6, 13},{3, 6, 14},{3, 6, 15},{3, 7, 0},{3, 7, 1},{3, 7, 1},{3, 7, 2},{3, 7, 3},{3, 7, 4},{3, 7, 5},{3, 7, 6},{3, 7, 7},{3, 7, 8},{3, 7, 9},{3, 7, 10},{3, 7, 11},{3, 7, 13},{3, 7, 14},{3, 7, 15},{3, 7, 16},{3, 7, 17},{3, 7, 18},{3, 7, 19},{3, 7, 20},{3, 7, 21},{3, 7, 22},{3, 7, 23},{3, 7, 24},{3, 7, 25},{3, 7, 26},{3, 7, 27},{3, 7, 28},{3, 7, 29},{3, 7, 30},{3, 7, 31},{3, 8, 0},{3, 9, 0},{3, 9, 1},{3, 9, 2},{3, 9, 3},{3, 10, 0},{3, 10, 1},{3, 10, 2},{3, 10, 3},{3, 10, 4},{3, 10, 5},{3, 10, 6},{3, 10, 7},{3, 10, 8},{3, 10, 9},{3, 10, 10},{3, 10, 11},{3, 10, 12},{3, 10, 13},{3, 10, 14},{3, 10, 15},{3, 11, 0},{3, 11, 1},{3, 11, 2},{3, 11, 3},{3, 11, 4},{3, 11, 5},{3, 11, 6},{3, 11, 7},{3, 11, 8},{3, 11, 9},{3, 11, 10},{3, 11, 11},{3, 11, 12},{3, 11, 13},{3, 11, 14},{3, 11, 15},{3, 12, 0},{3, 12, 1},{3, 12, 2},{3, 12, 3},{3, 13, 0},{3, 13, 1},{3, 13, 2},{3, 13, 3},{3, 13, 4},{3, 13, 5},{3, 13, 6},{3, 13, 7},{3, 13, 8},{3, 13, 9},{3, 13, 10},{3, 13, 11},{3, 13, 12},{3, 13, 13},{3, 13, 14},{3, 13, 15},{3, 14, 0},{3, 14, 1},{3, 14, 2},{3, 14, 3},{3, 15, 0},{3, 16, 0},{3, 16, 1},{3, 16, 2},{3, 16, 3},{3, 16, 4},{3, 16, 5},{3, 16, 6},{3, 16, 7},{3, 16, 8},{3, 16, 9},{3, 16, 10},{3, 16, 11},{3, 16, 12},{3, 16, 13},{3, 16, 14},{3, 16, 15},{3, 16, 16},{3, 16, 17},{3, 16, 18},{3, 16, 19},{3, 16, 20},{3, 16, 21},{3, 16, 22},{3, 16, 23},{3, 16, 24},{3, 16, 25},{3, 16, 26},{3, 16, 27},{3, 16, 28},{3, 16, 29},{3, 16, 30},{3, 16, 31},{3, 16, 32},{3, 16, 33},{3, 16, 34},{3, 16, 35},{3, 16, 36},{3, 16, 37},{3, 16, 38},{3, 16, 39},{3, 16, 40},{3, 16, 41},{3, 16, 42},{3, 16, 43},{3, 16, 44},{3, 16, 45},{3, 16, 46},{3, 16, 47},{3, 16, 48},{3, 16, 49},{3, 16, 50},{3, 16, 51},{3, 16, 52},{3, 16, 53},{3, 16, 54},{3, 16, 55},{3, 16, 56},{3, 16, 57},{3, 16, 58},{3, 16, 59},{3, 16, 60},{3, 16, 61},{3, 16, 62},{3, 16, 63},{3, 16, 64},{3, 16, 65},{3, 16, 66},{3, 16, 67},{3, 16, 68},{3, 16, 69},{3, 16, 70},{3, 16, 71},{3, 16, 72},{3, 16, 73},{3, 16, 74},{3, 16, 75},{3, 16, 76},{3, 16, 77},{3, 16, 78},{3, 16, 79},{3, 16, 80},{3, 16, 81},{3, 16, 82},{3, 16, 83},{3, 16, 84},{3, 16, 85},{3, 16, 86},{3, 16, 87},{3, 16, 88},{3, 16, 89},{3, 16, 90},{3, 16, 91},{3, 16, 92},{3, 16, 93},{3, 16, 94},{3, 16, 95},{3, 16, 96},{3, 16, 97},{3, 16, 98},{3, 16, 99},{3, 17, 0},{3, 17, 1},{3, 17, 2},{3, 17, 3},{3, 18, 0},{3, 18, 1},{3, 18, 2},{3, 18, 3},{3, 19, 0},{3, 19, 1},{3, 20, 0},{3, 20, 1},{3, 20, 2},{3, 20, 3},{3, 20, 4},{3, 20, 5},{3, 20, 6},{3, 20, 7},{3, 21, 0},{3, 21, 1},{3, 21, 2},{3, 21, 3},{3, 21, 4},{3, 21, 5},{3, 21, 6},{3, 21, 7},{3, 22, 0},{3, 22, 1},{3, 22, 2},{3, 22, 3},{3, 22, 4},{3, 22, 5},{3, 22, 6},{3, 22, 7},{3, 23, 0},{3, 23, 1},{3, 24, 0},{3, 24, 1},{3, 24, 2},{3, 24, 3},{3, 25, 0},{3, 25, 1},{3, 25, 2},{3, 25, 3},{3, 25, 4},{3, 25, 5},{3, 25, 6},{3, 25, 7},{3, 25, 8},{3, 25, 9},{3, 25, 10},{3, 25, 11},{3, 25, 12},{3, 25, 13},{3, 25, 14},{3, 25, 15},{3, 25, 16},{3, 25, 17},{3, 25, 18},{3, 25, 19},{3, 25, 20},{3, 25, 21},{3, 25, 22},{3, 25, 23},{3, 25, 24},{3, 25, 25},{3, 25, 26},{3, 25, 27},{3, 25, 28},{3, 25, 29},{3, 25, 30},{3, 25, 31},{3, 26, 0},{3, 26, 1},{3, 26, 2},{3, 26, 3},{3, 27, 0},{3, 27, 1},{3, 27, 2},{3, 27, 3},{3, 28, 0},{3, 28, 1},{3, 28, 2},{3, 28, 3},{3, 28, 4},{3, 28, 5},{3, 28, 6},{3, 28, 7},{3, 29, 0},{3, 29, 1},{3, 29, 2},{3, 29, 3},{3, 29, 4},{3, 29, 5},{3, 29, 6},{3, 29, 7},{3, 30, 0},{3, 30, 1},{3, 31, 0},{3, 31, 1},{3, 32, 0},{3, 33, 0},{3, 33, 1},{3, 33, 2},{3, 33, 3},{3, 34, 0},{3, 35, 0},{3, 35, 1},{3, 35, 2},{3, 35, 3},{3, 35, 4},{3, 35, 5},{3, 35, 6},{3, 35, 7},{3, 36, 0},{3, 37, 0},{3, 37, 1},{3, 38, 0},{3, 38, 1},{3, 38, 2},{3, 38, 3},{3, 39, 0},{3, 39, 1},{3, 39, 2},{3, 39, 3},{3, 39, 4},{3, 39, 5},{3, 39, 6},{3, 39, 7},{4, 0, 0},{4, 0, 1},{4, 1, 0},{4, 1, 1},{4, 1, 2},{4, 1, 3},{4, 1, 4},{4, 1, 5},{4, 1, 6},{4, 1, 7},{4, 1, 8},{4, 1, 9},{4, 1, 10},{4, 1, 11},{4, 1, 12},{4, 1, 13},{4, 1, 14},{4, 1, 15},{4, 2, 0},{4, 2, 1},{4, 3, 0},{4, 3, 1},{4, 4, 0},{4, 4, 1},{4, 4, 2},{4, 4, 3},{4, 5, 0},{4, 5, 1},{4, 5, 2},{4, 5, 3},{4, 5, 4},{4, 5, 5},{4, 5, 6},{4, 5, 7},{4, 6, 0},{4, 6, 1},{4, 6, 2},{4, 6, 3},{4, 7, 0},{4, 7, 1},{4, 7, 2},{4, 7, 3},{4, 7, 4},{4, 7, 5},{4, 7, 6},{4, 7, 7},{4, 7, 8},{4, 7, 9},{4, 7, 10},{4, 7, 11},{4, 7, 12},{4, 7, 13},{4, 7, 14},{4, 7, 15},{4, 8, 0},{4, 9, 0},{4, 9, 1},{4, 9, 2},{4, 9, 3},{4, 10, 0},{4, 10, 1},{4, 10, 2},{4, 10, 3},{4, 11, 0},{4, 11, 1},{4, 11, 2},{4, 11, 3},{4, 11, 4},{4, 11, 5},{4, 11, 6},{4, 11, 7},{4, 12, 0},{4, 12, 1},{4, 13, 0},{4, 13, 1},{4, 13, 2},{4, 13, 3},{4, 14, 0},{4, 14, 1},{4, 14, 2},{4, 14, 3},{4, 14, 4},{4, 14, 5},{4, 14, 6},{4, 14, 7},{4, 15, 0},{4, 16, 0},{4, 16, 1},{4, 16, 2},{4, 16, 3},{4, 16, 4},{4, 16, 5},{4, 16, 6},{4, 16, 7},{4, 16, 8},{4, 16, 9},{4, 16, 10},{4, 16, 11},{4, 16, 12},{4, 16, 13},{4, 16, 14},{4, 16, 15},{4, 16, 16},{4, 16, 17},{4, 16, 18},{4, 16, 19},{4, 16, 20},{4, 16, 21},{4, 16, 22},{4, 16, 23},{4, 16, 24},{4, 16, 25},{4, 16, 26},{4, 16, 27},{4, 16, 28},{4, 16, 29},{4, 16, 30},{4, 16, 31},{4, 16, 32},{4, 16, 33},{4, 16, 34},{4, 16, 35},{4, 16, 36},{4, 16, 37},{4, 16, 38},{4, 16, 39},{4, 16, 40},{4, 16, 41},{4, 16, 42},{4, 16, 43},{4, 16, 44},{4, 16, 45},{4, 16, 46},{4, 16, 47},{4, 16, 48},{4, 16, 49},{4, 16, 50},{4, 16, 51},{4, 16, 52},{4, 16, 53},{4, 16, 54},{4, 16, 55},{4, 16, 56},{4, 16, 57},{4, 16, 58},{4, 16, 59},{4, 16, 60},{4, 16, 61},{4, 16, 62},{4, 16, 63},{4, 17, 0},{4, 17, 1},{4, 18, 0},{4, 18, 1},{4, 19, 0},{4, 19, 1},{4, 20, 0},{4, 20, 1},{4, 20, 2},{4, 20, 3},{4, 21, 0},{4, 21, 1},{4, 21, 2},{4, 21, 3},{4, 22, 0},{4, 22, 1},{4, 22, 2},{4, 22, 3},{4, 22, 4},{4, 22, 5},{4, 22, 6},{4, 22, 7},{4, 23, 0},{4, 23, 1},{4, 24, 0},{4, 24, 1},{4, 25, 0},{4, 25, 1},{4, 25, 2},{4, 25, 3},{4, 25, 4},{4, 25, 5},{4, 25, 6},{4, 25, 7},{4, 26, 0},{4, 27, 0},{4, 28, 0},{4, 28, 1},{4, 28, 2},{4, 28, 3},{4, 29, 0},{4, 29, 1},{4, 29, 2},{4, 29, 3},{4, 29, 4},{4, 29, 5},{4, 29, 6},{4, 29, 7},{4, 29, 8},{4, 29, 9},{4, 29, 10},{4, 29, 11},{4, 29, 12},{4, 29, 13},{4, 29, 14},{4, 29, 15},{4, 30, 0},{4, 31, 0},{4, 32, 0},{4, 32, 1},{4, 32, 2},{4, 32, 3},{4, 32, 4},{4, 32, 5},{4, 32, 6},{4, 32, 7},{4, 33, 0},{4, 33, 1},{4, 34, 0},{4, 35, 0},{4, 35, 1},{4, 35, 2},{4, 35, 3},{4, 35, 4},{4, 35, 5},{4, 35, 6},{4, 35, 7},{4, 35, 8},{4, 35, 9},{4, 35, 10},{4, 35, 11},{4, 35, 12},{4, 35, 13},{4, 35, 14},{4, 35, 15},{4, 36, 0},{4, 37, 0},{4, 37, 1},{4, 38, 0},{4, 38, 1},{4, 39, 0},{4, 39, 1},{5, 0, 0},{5, 0, 1},{5, 1, 0},{5, 2, 0},{5, 2, 1},{5, 3, 0},{5, 4, 0},{5, 5, 0},{5, 5, 1},{5, 5, 2},{5, 5, 3},{5, 5, 4},{5, 5, 5},{5, 5, 6},{5, 5, 7},{5, 6, 0},{5, 7, 0},{5, 8, 0},{5, 9, 0},{5, 9, 1},{5, 10, 0},{5, 11, 0},{5, 12, 0},{5, 13, 0},{5, 14, 0},{5, 14, 1},{5, 14, 2},{5, 14, 3},{5, 15, 0},{5, 16, 0},{5, 17, 0},{5, 18, 0},{5, 19, 0},{5, 19, 1},{5, 19, 2},{5, 19, 3},{5, 20, 0},{5, 21, 0},{5, 22, 0},{5, 23, 0},{5, 23, 1},{5, 24, 0},{5, 25, 0},{5, 26, 0},{5, 27, 0},{5, 28, 0},{5, 29, 0},{5, 29, 1},{5, 29, 2},{5, 29, 3},{5, 30, 0},{5, 31, 0},{5, 32, 0},{5, 32, 1},{5, 32, 2},{5, 32, 3},{5, 32, 4},{5, 32, 5},{5, 32, 6},{5, 32, 7},{5, 33, 0},{5, 34, 0},{5, 35, 0},{5, 35, 1},{5, 35, 2},{5, 35, 3},{5, 36, 0},{5, 37, 0},{5, 37, 1},{5, 37, 2},{5, 37, 3},{5, 38, 0},{5, 39, 0}};

  vector<vector<vector<unsigned int>>> designPoints;

  for(int i = 0; i<1047; i++)
  {
    string designPointFileString = "data/arch";
    designPointFileString += std::to_string(fileInfoArray[i][0]);
    designPointFileString += "_app";
    designPointFileString += std::to_string(fileInfoArray[i][1]);
    designPointFileString += "_bind";
    designPointFileString += std::to_string(fileInfoArray[i][2]);
    designPointFileString += ".txt";
    std::fstream designPointFile(designPointFileString);
    GotoLine(designPointFile, 6);
    string line;
    vector<vector<unsigned int>> designPoint;
    while(std::getline(designPointFile, line))
    {
      std::istringstream ss(line);
      unsigned int inputSize;
      vector<unsigned int> functionInputs;
      while(ss >> inputSize)
      {
        functionInputs.push_back(inputSize);
      }
      designPoint.push_back(functionInputs);
    }
    designPoints.push_back(designPoint);
  }

  vector<vector<unsigned int>> archVector;
  vector<unsigned int> allocatedPEs;
  for(int i = 0; i<6; i++)
  {
    string archFileString = "data/arch";
    archFileString += std::to_string(i);
    archFileString += ".txt";
    std::fstream archFile(archFileString);
    
    GotoLine(archFile, 6);
    string line;
    std::getline(archFile, line);
    unsigned int allocatedPe;
    std::istringstream ss(line);
    while(ss >> allocatedPe)
    {
      allocatedPEs.push_back(allocatedPe);
    }
    archVector.push_back(allocatedPEs);
    allocatedPEs.clear();
  }

  designPointsCount = 1047;
  peCount = 37;
  funcCount = 38;

  float *designSpaceTensor;
  unsigned int designSpaceSize = designPointsCount*peCount*funcCount;

  hipMallocManaged(&designSpaceTensor, designSpaceSize*sizeof(float));
  
  unsigned int threadCount = 1024;
  unsigned int blockCount = (designSpaceSize+(threadCount-1))/threadCount;

  v_set KERNEL_ARG2(blockCount,threadCount)(designSpaceTensor, 0, designSpaceSize);
  hipDeviceSynchronize();

  for(auto dpIndex = 0; dpIndex < designPoints.size(); dpIndex++)
  {
    for(auto funcTypeIndex = 0; funcTypeIndex < designPoints[dpIndex].size(); funcTypeIndex++)
    {
      for(auto peIndex = 0; peIndex < designPoints[dpIndex][funcTypeIndex].size(); peIndex++)
      {
        unsigned int peInputSize = designPoints[dpIndex][funcTypeIndex][peIndex];
        unsigned int tensorPeIndex = archVector[fileInfoArray[dpIndex][0]][peIndex];
        designSpaceTensor[dpIndex*(peCount*funcCount) + funcTypeIndex*peCount + tensorPeIndex] = peInputSize;
      } 
    }
  }

  return designSpaceTensor;
}

float* loadPerfTable(unsigned int& coefficientCount, unsigned int peCount, unsigned int funcCount)
{
  float c0[1406] = {0, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, 15, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, 25, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, 25, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, 45, 45, 50, 50, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 835, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 15, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 15, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 75, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, 45, 45, 50, 50, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, 50, 50, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 25, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 105, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 25, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, 45, 45, 50, 50, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 12.5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 155, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 15, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 12.5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 140, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 31.5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 345, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 25, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 225, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 200, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 105, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 50, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 25, 25, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 105, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 570, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, 50, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 45, -1, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 120, -1, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 50, -1, -1, -1, -1, -1, -1, 45 };
  float c1[1406] = {0.5, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 2, -1, 20, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0.5, -1, -1, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 3, -1, -1, -1, 30, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 5, -1, -1, -1, -1, 30, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0.5, -1, -1, -1, -1, -1, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 40, -1, -1, -1, -1, -1, -1, 50, 50, 55, 55, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 314, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 840, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 20, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 20, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 39, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 80, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 40, -1, -1, -1, -1, -1, -1, 50, 50, 55, 55, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 112, -1, -1, -1, -1, -1, -1, -1, -1, 55, 55, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 22, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 30, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 110, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 22, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 30, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 40, -1, -1, -1, -1, -1, -1, 50, 50, 55, 55, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 28, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 13.75, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 570, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 160, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 3.5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 20, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 28, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 13.75, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 16.5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 145, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 35, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 355, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 26, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 30, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0.5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 671.5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 230, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 20.5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 205, -1, -1, -1, -1, -1, -1, -1, -1, -1, 5.5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 106.25, -1, -1, -1, -1, -1, -1, -1, -1, 224, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 55, -1, -1, -1, -1, -1, -1, -1, 0.5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 5, -1, -1, -1, -1, -1, -1, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 30, 30, -1, -1, -1, -1, 31, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 110, -1, -1, -1, -1, 57.5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 575, -1, -1, -1, 112, -1, -1, -1, -1, -1, -1, -1, -1, -1, 55, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 50, -1, -1, 31.5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 125, -1, 80, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 55, -1, -1, -1, -1, -1, -1, 50};

  float* perfTable;
  coefficientCount = 2;
  unsigned int perfTableSize = peCount*funcCount;
  hipMallocManaged(&perfTable, perfTableSize*sizeof(float));

  unsigned int threadCount = 1024;
  unsigned int blockCount = (perfTableSize+(threadCount-1))/threadCount;

  v_set KERNEL_ARG2(blockCount,threadCount)(perfTable, 0, perfTableSize);
  hipDeviceSynchronize();

  for(int i = 0; i<peCount*funcCount; i++)
  {
    perfTable[i] = c0[i];
  }

  for(int i = 0; i<peCount*funcCount; i++)
  {
    perfTable[peCount*funcCount + i] = c1[i];
  }

  // for(int i = 0; i<2; i++)
  // {
  //   printf("c[%d]\n", i);
  //   for(int j = 0; j<38; j++)
  //   {
  //     for(int k = 0; k<37; k++)
  //     {
    
  //       printf("%.02f ", perfTable[i*peCount*funcCount + j*peCount + k]);
  //     }
  //     printf("\n");
  //   }
  // }

  return perfTable;
}


// Kernel function to add the elements of two arrays
__global__ void t_mult(float *designSpaceTensor, float *perfTable, float* latencyTensor, int designPointsCount,int peCount,int funcCount)
{
  int designPointSize = peCount*funcCount;
  int thrdDesignPointIndex = blockIdx.x * blockDim.x + threadIdx.x;
  int perfTableC1Offset = designPointSize;
  int globalIndex = blockIdx.y*designPointSize + thrdDesignPointIndex;
  int stride = peCount;
  if(thrdDesignPointIndex < peCount)
  {
    for (int dpIdx = globalIndex, perfIdx = thrdDesignPointIndex; perfIdx < designPointSize; dpIdx += stride, perfIdx += stride)
    {
        latencyTensor[dpIdx] = perfTable[perfIdx] + designSpaceTensor[dpIdx]*perfTable[perfIdx + perfTableC1Offset];
        // if(threadIdx.x == 0)
        // {
        //   printf("G %d, dpIdx %d\n",globalIndex, dpIdx);
        // }
        // __syncthreads();
    }
  }
}

float* createAndSetCudaManagedMemory(unsigned int size)
{
  float* mem; 
  hipMallocManaged(&mem, size*sizeof(float));

  unsigned int threadCount = 1024;
  unsigned int blockCount = (size+(threadCount-1))/threadCount;

  v_set KERNEL_ARG2(blockCount,threadCount)(mem, 0, size);
  hipDeviceSynchronize();

  return mem;
}

int main(void)
{
  
  unsigned int designPointsCount;
  unsigned int peCount;
  unsigned int funcCount;
  unsigned int coefficientCount;

  float* designSpaceTensor = loadDSTensor(designPointsCount, peCount, funcCount);
  float* perfTable = loadPerfTable(coefficientCount, peCount, funcCount);

  unsigned int designSpaceSize = designPointsCount*peCount*funcCount;

  float* latencyTensor = createAndSetCudaManagedMemory(designSpaceSize);
  
  dim3 dimGrid;
  unsigned int threadCount = 32;
  dimGrid.x = (peCount+(threadCount-1))/threadCount;
  dimGrid.y = designPointsCount;

  t_mult KERNEL_ARG2(dimGrid,threadCount)(designSpaceTensor,perfTable,latencyTensor,designPointsCount,peCount,funcCount);
  hipDeviceSynchronize();


  // auto graph = loadcoAuthors();
  // int m,n;

  // gettimeofday(&t1, 0);

  // unsigned int* arr = flattenGraph(graph,m,n);
  // unsigned int* coAuthorSums = NULL;
  // auto rankings = rankAuthorByCoAuthorCount(arr, &coAuthorSums, m, n);
  // unsigned int maxCoAuthorCount = rankings.begin()->second + 1; // 0 coauthors 
  // auto histogram = coAuthorCountHistogram(coAuthorSums, m, n, maxCoAuthorCount);

  // gettimeofday(&t2, 0);

  // int count = 0; 
  // printf("TOP 10 AUTHORS!\n");
  // for(auto author = rankings.begin(); count < 10; author++)
  // {
  //     printf("AUTHOR[%d] CoAuthorCount %d\n", author->first, author->second);
  //     count++;
  // }
  // printf("COAUTHOR HISTOGRAM!\n");
  // for(auto bin : histogram)
  // {
  //   printf("bin[%d] = %d\n", bin.first, bin.second);
  // }


  // printf("Authors that share a coAuthor Count %d\n", getAuthorThatShareCoAuthorCount(arr,m, n));

  // // Free memory
  // hipFree(arr);
  // hipFree(coAuthorSums);

  // double time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;

  // printf("Time to Evaluate:  %3.1f ms \n", time);

  return 0;
}
